#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <cmath>
// #include a timer module

hipblasHandle_t handler;


// Multiplication instances
void GPU_multi(double *A, double *B, double *C, int lda, int ldb, int ldc, int xa, int xb, int xc, int ya, int yb, int yc, double aleph, double bet) {
	hipblasDgemm(handler, HIPBLAS_OP_N, HIPBLAS_OP_N, xb, ya, xa, &aleph, B, ldb, A, lda, &bet, C, ldc);
}

